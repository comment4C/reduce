
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<math.h>

#define N 1024

__global__ void interleaved_reduce(int* d_in, int* d_out) {
	int i = threadIdx.x;

	__shared__ int sB[N];
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	sB[i] = d_in[id];
	__syncthreads();

	for(int s = 1; s < blockDim.x; s = s*2) {
		int index = 2 * s * id;
		if(index < blockDim.x) {
			sB[index] += sB[index+s];
		}
		__syncthreads();
	}
	if(i == 0)
		d_out[blockIdx.x] = sB[0];
}

__global__ void contiguous_reduce(int* d_in, int* d_out) {
	int i = threadIdx.x;
    int M = N/2;
    for(int s = M; s > 0; s=s>>1) {
        if(i < M) {
            d_in[i] = d_in[i] + d_in[i+s];
        }
        M = M/2;
    }
    if(i == 0)
        d_out[0] = d_in[0];
}


int main() {
	int h_in[N];
	int h_out;

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	for(int i = 0; i < N; i++) {
		h_in[i] = i+1;	
	}
	
	int *d_in, *d_out;

	//Part 1: Memory transfer from host to device
	hipMalloc((void**) &d_in, N*sizeof(int));
	hipMalloc((void**) &d_out, sizeof(int));

	hipMemcpy(d_in, &h_in, N*sizeof(int), hipMemcpyHostToDevice);

	//Part 2: Execute kernel
	
	hipEventRecord(start);
    // interleaved_reduce<<<1, 1024>>>(d_in, d_out);
    contiguous_reduce<<<1, 1024>>>(d_in, d_out);
	hipEventRecord(stop);

	//Part 3: Memory transfer from device to host
	hipMemcpy(&h_out, d_out, sizeof(int), hipMemcpyDeviceToHost);

	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);

	hipFree(d_in);
	hipFree(d_out);

	printf("Output: %d\n", h_out);
	printf("%f milliseconds\n", milliseconds);

	return -1;
}